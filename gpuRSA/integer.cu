#include "hip/hip_runtime.h"
#include "integer.h"

#include <string.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void cuda_crackKeys(const integer *keys, uint16_t *block_noCoprime, int gridRow, int gridCol, int gridDim, int keyNum) {

  // In each block, we use two shared arrays to record the key pairs
  __shared__ volatile uint32_t keyOne[BLOCK_DIM][BLOCK_DIM][32];
  __shared__ volatile uint32_t keyTwo[BLOCK_DIM][BLOCK_DIM][32];

  // This will generate two keys for this block to compare.
  int keyX = gridCol * gridDim + blockIdx.x * BLOCK_DIM + threadIdx.x;
  int keyY = gridRow * gridDim + blockIdx.y * BLOCK_DIM + threadIdx.y;

  //We only need to compare each pair of key for one time
  if (keyX < keyNum && keyY < keyNum && keyX > keyY) {
    //Each thread will load its corresponding chunk
    keyOne[threadIdx.x][threadIdx.y][threadIdx.z] = keys[keyX].ints[threadIdx.z];
    keyTwo[threadIdx.x][threadIdx.y][threadIdx.z] = keys[keyY].ints[threadIdx.z];

    //Calculate gcd for each pair of keys
    gcd(keyOne[threadIdx.x][threadIdx.y], keyTwo[threadIdx.x][threadIdx.y]);

    if (threadIdx.x == 31) {
      // If gcd > 1, it means the pair is coPrime, and we need to record it.
      if ((keyTwo[threadIdx.x][threadIdx.y][threadIdx.z]) > 1) {
        int noCoprimeBlockId = blockIdx.y * gridDim.x + blockIdx.x;
        block_noCoprime[noCoprimeBlockId] |= 1 << threadIdx.y * BLOCK_DIM + threadIdx.x;
      }
    }
  }
}

/**
 * Binary GCD algo
 */
__device__ void gcd(volatile uint32_t *x, volatile uint32_t *y) {
  int tid = threadIdx.z;

  while (__any(x[tid])) {
    while ((x[31] & 1) == 0)
      shiftR1(x);

    while ((y[31] & 1) == 0)
      shiftR1(y);

    if (geq(x, y)) {
      cuSubtract(x, y, x);
      shiftR1(x);
    }
    else {
      cuSubtract(y, x, y);
      shiftR1(y);
    }
  }
}

__device__ void shiftR1(volatile uint32_t *x) {
  int tid = threadIdx.z;
  uint32_t prevX = tid ? x[tid-1] : 0;
  x[tid] = (x[tid] >> 1) | (prevX << 31);
}

__device__ void shiftL1(volatile uint32_t *x) {
  int tid = threadIdx.z;
  uint32_t nextX = tid != 31 ? x[tid+1] : 0;
  x[tid] = (x[tid] << 1) | (nextX >> 31);
}

__device__ int geq(volatile uint32_t *x, volatile uint32_t *y) {
  /* shared memory to hold the position at which the int of x >= int of y */
  __shared__ unsigned int pos[BLOCK_DIM][BLOCK_DIM];
  int tid = threadIdx.z;

  if (tid == 0)
    pos[threadIdx.x][threadIdx.y] = 31;

  if (x[tid] != y[tid])
    atomicMin(&pos[threadIdx.x][threadIdx.y], tid);

  return x[pos[threadIdx.x][threadIdx.y]] >= y[pos[threadIdx.x][threadIdx.y]];
}

__device__ void cuSubtract(volatile uint32_t *x, volatile uint32_t *y, volatile uint32_t *z) {
  /* shared memory to hold underflow flags */
  __shared__ unsigned char s_borrow[BLOCK_DIM][BLOCK_DIM][32];
  unsigned char *borrow = s_borrow[threadIdx.x][threadIdx.y];
  int tid = threadIdx.z;

  /* set LSB's borrow to 0 */
  if (tid == 0)
    borrow[31] = 0;

  uint32_t t;
  t = x[tid] - y[tid];

  /* set the previous int's underflow flag if the subtraction answer is bigger than the subtractee */
  if(tid)
    borrow[tid - 1] = (t > x[tid]);

  /* keep processing until there's no flags */
  while (__any(borrow[tid])) {
    if (borrow[tid])
      t--;

    /* have to set flag if the new sub answer is 0xFFFFFFFF becuase of an underflow */
    if (tid)
      borrow[tid - 1] = (t == 0xFFFFFFFFu && borrow[tid]);
  }

  z[tid] = t;
}
